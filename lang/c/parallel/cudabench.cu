#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cudabench.h"

#define assertCudaSuccess(ans) { _assertCudaSuccess((ans), __FILE__, __LINE__); }
inline void _assertCudaSuccess(hipError_t code, char *file, int line)
{
  if (code != hipSuccess) {
    fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

// dcopy
__global__ void
_dcopy_cuda (const double * S, double * D, size_t length)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < length) D[id] = S[id];
}
void
dcopy_cuda (const double * A, double * B, size_t length)
{
  size_t size = sizeof(double) * length;
  // malloc
  double * d_A = NULL, * d_B = NULL;
  hipMalloc ((void**)&d_A, size);
  hipMalloc ((void**)&d_B, size);
  // transter H -> D
  hipMemcpy (d_A, A, size, hipMemcpyHostToDevice);
  // apply kernel
  int threadsperblock = 256;
  int blockspergrid = (length + threadsperblock - 1)/threadsperblock;
  _dcopy_cuda <<<blockspergrid, threadsperblock>>> (d_A, d_B, length);
  // transter D -> H
  hipMemcpy (B, d_B, size, hipMemcpyDeviceToHost);
  // free
  hipFree (d_A);
  hipFree (d_B);
}

// dscal
__global__ void
_dscal_cuda (double * x, const double a, size_t n)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < n) x[id] = x[id] * a;
}
void
dscal_cuda (double * x, const double a, size_t n)
{
  size_t size = sizeof(double) * n;
  // malloc
  double * d_A = NULL;
  hipMalloc ((void**)&d_A, size);
  // transter H -> D
  hipMemcpy (d_A, x, size, hipMemcpyHostToDevice);
  // apply kernel
  int threadsperblock = 256;
  int blockspergrid = (n + threadsperblock - 1)/threadsperblock;
  _dscal_cuda <<<blockspergrid, threadsperblock>>> (d_A, a, n);
  // transter D -> H
  hipMemcpy (x, d_A, size, hipMemcpyDeviceToHost);
  // free
  hipFree (d_A);
}

// dasum
__global__ void
_dasum_cuda (double * a, size_t n, double * o)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  //extern __shared__ double sd[];
  //if (id < n) sd[threadIdx.x] = a[id];
  //__syncthreads();

  for (size_t b=(blockDim.x / 2); b > 0; b >>= 1) {
    if (threadIdx.x < b && id < n) {
      a[id] += a[id+b];
      //sd[threadIdx.x] += sd[threadIdx.x + b];
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    o[blockIdx.x] = a[id];
    //o[blockIdx.x] = sd[0];
  }
}
double
dasum_cuda (const double * a, size_t n)
{
  double sum = 0.;
  size_t size = sizeof(double) * n;
  int threadsperblock = 256;
  int blockspergrid = (n + threadsperblock - 1)/threadsperblock;
  if (blockspergrid == 0) blockspergrid = 1; // fixes bug when vector is short.
  printf ("<<<%ld, %ld>>>\n", blockspergrid, threadsperblock);
  // malloc
  double * d_A = NULL, * d_S = NULL;
  hipMalloc ((void **)&d_A, size);
  hipMalloc ((void **)&d_S, blockspergrid*sizeof(double));
  // transter H -> D
  hipMemcpy (d_A, a, size, hipMemcpyHostToDevice);
  hipMemset (d_S, 0, blockspergrid*sizeof(double));
  // apply kernel
  _dasum_cuda <<<blockspergrid, threadsperblock>>> (d_A, n, d_S);
  // transfer D -> H
  double * h_S = NULL;
  h_S = (double *) malloc (blockspergrid*sizeof(double));
  memset ((void *)h_S, 0, blockspergrid*sizeof(double));
  hipMemcpy (h_S, d_S, blockspergrid*sizeof(double), hipMemcpyDeviceToHost);
  for (size_t i = 0; i < blockspergrid; i++) {
    printf ("%lf (%ld, %ld)\n", h_S[i], i, blockspergrid);
  }
  for (size_t i = 0; i < blockspergrid; i++) sum += h_S[i];
  free (h_S);
  hipFree (d_A);
  hipFree (d_S);
  return sum;
}
